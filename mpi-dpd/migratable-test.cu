#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

#include <mpi.h>

#include "migration.h"
#include "migratable-datastructures.h"

class MigratableThing : public Migratable<20>
{
public:
    explicit MigratableThing() : n_ptrs(0), buffer_size(512), id(id),
        dev_buffer2(this), host_buffer2(this), pin_buffer2(this) {}

    void alloc_some_data(int type)
    {
        assert(type >= 0 && type < 4);
        if (n_ptrs >= MAX_BUFFERS - 4)
            return;
        switch (type) {
        case 0:
            malloc_migratable(&ptrs[n_ptrs], buffer_size);
            on_device[n_ptrs++] = false;
            break;
        case 1:
            malloc_migratable_device(&ptrs[n_ptrs], buffer_size);
            on_device[n_ptrs++] = true;
            break;
        case 2:
            malloc_migratable_host(&ptrs[n_ptrs], buffer_size);
            on_device[n_ptrs++] = false;
            break;
        case 3:
            malloc_migratable_pinned(&ptrs[n_ptrs], buffer_size);
            on_device[n_ptrs++] = false;
            break;
        }
    }

    void do_some_stuff()
    {
        for (int i = 0; i < n_ptrs; ++i) {
            if (on_device[i])
                hipMemset(ptrs[i], 0, buffer_size);
            else
                memset(ptrs[i], 0, buffer_size);
        }
    }

    void free_last()
    {
        if (n_ptrs == 0)
            return;
        free_migratable(ptrs[--n_ptrs]);
    }

    void resize_device_buffer(int size)
    {
        dev_buffer.resize(size);
    }

    void preserve_resize_device_buffer(int size)
    {
        dev_buffer.preserve_resize(size);
    }

    void resize_host_buffer(int size)
    {
        host_buffer.resize(size);
    }

    void preserve_resize_host_buffer(int size)
    {
        host_buffer.preserve_resize(size);
    }

    void resize_pinned_buffer(int size)
    {
        pin_buffer.resize(size);
    }

    void preserve_resize_pinned_buffer(int size)
    {
        pin_buffer.preserve_resize(size);
    }

    void resize_device_buffer2(int size)
    {
        dev_buffer2.resize(size);
    }

    void preserve_resize_device_buffer2(int size)
    {
        dev_buffer2.preserve_resize(size);
    }

    void resize_host_buffer2(int size)
    {
        host_buffer2.resize(size);
    }

    void preserve_resize_host_buffer2(int size)
    {
        host_buffer2.preserve_resize(size);
    }

    void resize_pinned_buffer2(int size)
    {
        pin_buffer2.resize(size);
    }

    void preserve_resize_pinned_buffer2(int size)
    {
        pin_buffer2.preserve_resize(size);
    }

    void set_buffer_size(int new_buffer_size) { buffer_size = new_buffer_size; }
    void set_id(int new_id) { id = new_id; }
    int get_id() const { return id; }
private:
    void* ptrs[MAX_BUFFERS];
    bool on_device[MAX_BUFFERS];
    int n_ptrs, buffer_size, id;

    MigratableDeviceBuffer<int> dev_buffer;
    MigratableHostBuffer<float> host_buffer;
    MigratablePinnedBuffer<double> pin_buffer;

    MigratableDeviceBuffer2<int> dev_buffer2;
    MigratableHostBuffer2<float> host_buffer2;
    MigratablePinnedBuffer2<double> pin_buffer2;
};

int main(int argc, char** argv)
{
    MPI_Init(&argc, &argv);

    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    srand(2342347 * rank);

    const int n = 1;
    MigratableThing things[n];
    for (int i = 0; i < n; ++i) {
        things[i].set_buffer_size(rand() % 1024);
        things[i].set_id(i + rank * n);
    }

    for (int i = 0; i < 100; ++i)  {
        char processor_name[MPI_MAX_PROCESSOR_NAME];
        int processor_name_length;
        MPI_Get_processor_name(processor_name, &processor_name_length);
        processor_name[processor_name_length + 1] = 0;
        std::cout << processor_name << ": ";

        int j = rand() % n;
        MigratableThing& mt = things[j];
        std::cout << "things[" << mt.get_id() << "]";
        int t = rand() % 14;
        int size = 0;
        switch (t) {
        case 0:
        case 1:
        case 2:
        case 3:
            mt.alloc_some_data(t);
            std::cout << ".alloc_some_data(" << t << ")";
            break;
        case 4:
            mt.free_last();
            std::cout << ".free_last()";
            break;
        case 5:
            size = rand() % 1024;
            mt.resize_device_buffer(size);
            std::cout << ".resize_device_buffer(" << size << ")";
            break;
        case 6:
            size = rand() % 1024;
            mt.preserve_resize_device_buffer(size);
            std::cout << ".preserve_resize_device_buffer(" << size << ")";
            break;
        case 7:
            size = rand() % 1024;
            mt.resize_host_buffer(size);
            std::cout << ".resize_host_buffer(" << size << ")";
            break;
        case 8:
            size = rand() % 1024;
            mt.preserve_resize_host_buffer(size);
            std::cout << ".preserve_resize_host_buffer(" << size << ")";
            break;
        case 9:
            size = rand() % 1024;
            mt.resize_pinned_buffer(size);
            std::cout << ".resize_pinned_buffer(" << size << ")";
            break;
        case 10:
            size = rand() % 1024;
            mt.preserve_resize_pinned_buffer(size);
            std::cout << ".preserve_resize_pinned_buffer(" << size << ")";
            break;
        case 11:
            size = rand() % 1024;
            mt.preserve_resize_host_buffer2(size);
            std::cout << ".preserve_resize_host_buffer2(" << size << ")";
            break;
        case 12:
            size = rand() % 1024;
            mt.resize_pinned_buffer2(size);
            std::cout << ".resize_pinned_buffer2(" << size << ")";
            break;
        case 13:
            size = rand() % 1024;
            mt.preserve_resize_pinned_buffer2(size);
            std::cout << ".preserve_resize_pinned_buffer2(" << size << ")";
            break;
        }
        std::cout << std::endl;

        for (int j = 0; j < n; ++j)
            things[j].do_some_stuff();

#ifdef AMPI
        MPI_Migrate();
        MPI_Barrier(MPI_COMM_WORLD);
        if (rank == 0)
            std::cout << "--- MIGRATION ---" << std::endl;
        MPI_Barrier(MPI_COMM_WORLD);
#endif
    }
    

    MPI_Finalize();
    return 0;
};
