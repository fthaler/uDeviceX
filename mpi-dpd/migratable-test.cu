#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

#include <mpi.h>

#include "migration.h"
#include "migratable-datastructures.h"

class MigratableThing : public Migratable<20, 0, 10>
{
public:
    explicit MigratableThing() : n_ptrs(0), buffer_size(512), id(id),
        dev_buffer2(this), host_buffer2(this), pin_buffer2(this) {}

    void alloc_some_data(int type)
    {
        assert(type >= 0 && type < 4);
        if (n_ptrs >= MAX_BUFFERS - 4)
            return;
        switch (type) {
        case 0:
            malloc_migratable(&ptrs[n_ptrs], buffer_size);
            on_device[n_ptrs++] = false;
            break;
        case 1:
            malloc_migratable_device(&ptrs[n_ptrs], buffer_size);
            on_device[n_ptrs++] = true;
            break;
        case 2:
            malloc_migratable_host(&ptrs[n_ptrs], buffer_size);
            on_device[n_ptrs++] = false;
            break;
        case 3:
            malloc_migratable_pinned(&ptrs[n_ptrs], buffer_size);
            on_device[n_ptrs++] = false;
            break;
        }
    }

    void alloc_some_array()
    {
        if (n_arrays >= MAX_ARRAYS)
            return;
        hipChannelFormatDesc d = hipCreateChannelDesc<float>();
        malloc_migratable_array(&arrays[n_arrays++], &d, make_hipExtent(41, 23, 12));
    }

    void do_some_stuff()
    {
        for (int i = 0; i < n_ptrs; ++i) {
            if (on_device[i])
                hipMemset(ptrs[i], 0, buffer_size);
            else
                memset(ptrs[i], 0, buffer_size);
        }
    }

    void do_some_array_stuff()
    {
        void* zeros = malloc(41 * 23 * 12 * sizeof(float));
        memset(zeros, 0, 14 * 23 * 12 * sizeof(float));
        for (int i = 0; i < n_arrays; ++i) {
            hipMemcpy3DParms p = {0};
            p.srcPtr = make_hipPitchedPtr(zeros, 41 * sizeof(float), 41, 23);
            p.dstArray = arrays[i];
            p.extent = make_hipExtent(41, 23, 12);
            p.kind = hipMemcpyHostToDevice;
            CUDA_CHECK(hipMemcpy3D(&p));
        }
        free(zeros);
    }

    void free_last()
    {
        if (n_ptrs == 0)
            return;
        free_migratable(ptrs[--n_ptrs]);
    }

    void free_last_array()
    {
        if (n_arrays == 0)
            return;
        free_migratable_array(arrays[--n_arrays]);
    }

    void resize_device_buffer(int size)
    {
        dev_buffer.resize(size);
    }

    void preserve_resize_device_buffer(int size)
    {
        dev_buffer.preserve_resize(size);
    }

    void resize_host_buffer(int size)
    {
        host_buffer.resize(size);
    }

    void preserve_resize_host_buffer(int size)
    {
        host_buffer.preserve_resize(size);
    }

    void resize_pinned_buffer(int size)
    {
        pin_buffer.resize(size);
    }

    void preserve_resize_pinned_buffer(int size)
    {
        pin_buffer.preserve_resize(size);
    }

    void resize_device_buffer2(int size)
    {
        dev_buffer2.resize(size);
    }

    void preserve_resize_device_buffer2(int size)
    {
        dev_buffer2.preserve_resize(size);
    }

    void resize_host_buffer2(int size)
    {
        host_buffer2.resize(size);
    }

    void preserve_resize_host_buffer2(int size)
    {
        host_buffer2.preserve_resize(size);
    }

    void resize_pinned_buffer2(int size)
    {
        pin_buffer2.resize(size);
    }

    void preserve_resize_pinned_buffer2(int size)
    {
        pin_buffer2.preserve_resize(size);
    }

    void set_buffer_size(int new_buffer_size) { buffer_size = new_buffer_size; }
    void set_id(int new_id) { id = new_id; }
    int get_id() const { return id; }
private:
    void* ptrs[MAX_BUFFERS];
    bool on_device[MAX_BUFFERS];
    int n_ptrs, buffer_size, id;

    hipArray_t arrays[MAX_ARRAYS];
    int n_arrays;

    MigratableDeviceBuffer<int> dev_buffer;
    MigratableHostBuffer<float> host_buffer;
    MigratablePinnedBuffer<double> pin_buffer;

    MigratableDeviceBuffer2<int> dev_buffer2;
    MigratableHostBuffer2<float> host_buffer2;
    MigratablePinnedBuffer2<double> pin_buffer2;
};

int main(int argc, char** argv)
{
    MPI_Init(&argc, &argv);

    int rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    srand(2342347 * rank);

    const int n = 1;
    MigratableThing things[n];
    for (int i = 0; i < n; ++i) {
        things[i].set_buffer_size(rand() % 1024);
        things[i].set_id(i + rank * n);
    }

    for (int i = 0; i < 100; ++i)  {
        char processor_name[MPI_MAX_PROCESSOR_NAME];
        int processor_name_length;
        MPI_Get_processor_name(processor_name, &processor_name_length);
        processor_name[processor_name_length + 1] = 0;
        std::cout << processor_name << ": ";

        int j = rand() % n;
        MigratableThing& mt = things[j];
        std::cout << "things[" << mt.get_id() << "]";
        int t = rand() % 17;
        int size = 0;
        switch (t) {
        case 0:
        case 1:
        case 2:
        case 3:
            mt.alloc_some_data(t);
            std::cout << ".alloc_some_data(" << t << ")";
            break;
        case 4:
            mt.free_last();
            std::cout << ".free_last()";
            break;
        case 5:
            size = rand() % 1024;
            mt.resize_device_buffer(size);
            std::cout << ".resize_device_buffer(" << size << ")";
            break;
        case 6:
            size = rand() % 1024;
            mt.preserve_resize_device_buffer(size);
            std::cout << ".preserve_resize_device_buffer(" << size << ")";
            break;
        case 7:
            size = rand() % 1024;
            mt.resize_host_buffer(size);
            std::cout << ".resize_host_buffer(" << size << ")";
            break;
        case 8:
            size = rand() % 1024;
            mt.preserve_resize_host_buffer(size);
            std::cout << ".preserve_resize_host_buffer(" << size << ")";
            break;
        case 9:
            size = rand() % 1024;
            mt.resize_pinned_buffer(size);
            std::cout << ".resize_pinned_buffer(" << size << ")";
            break;
        case 10:
            size = rand() % 1024;
            mt.preserve_resize_pinned_buffer(size);
            std::cout << ".preserve_resize_pinned_buffer(" << size << ")";
            break;
        case 11:
            size = rand() % 1024;
            mt.preserve_resize_host_buffer2(size);
            std::cout << ".preserve_resize_host_buffer2(" << size << ")";
            break;
        case 12:
            size = rand() % 1024;
            mt.resize_pinned_buffer2(size);
            std::cout << ".resize_pinned_buffer2(" << size << ")";
            break;
        case 13:
            size = rand() % 1024;
            mt.preserve_resize_pinned_buffer2(size);
            std::cout << ".preserve_resize_pinned_buffer2(" << size << ")";
            break;
        case 14:
            mt.alloc_some_array();
            std::cout << ".alloc_some_array()";
            break;
        case 15:
            mt.free_last_array();
            std::cout << ".free_last_array()";
            break;
        case 16:
            mt.do_some_array_stuff();
            std::cout << ".do_some_array_stuff()";
            break;
        default:
            assert(0);
        }
        std::cout << std::endl;

        for (int j = 0; j < n; ++j)
            things[j].do_some_stuff();

#ifdef AMPI
        MPI_Migrate();
        MPI_Barrier(MPI_COMM_WORLD);
        if (rank == 0)
            std::cout << "--- MIGRATION ---" << std::endl;
        MPI_Barrier(MPI_COMM_WORLD);
#endif
    }
    

    MPI_Finalize();
    return 0;
};
